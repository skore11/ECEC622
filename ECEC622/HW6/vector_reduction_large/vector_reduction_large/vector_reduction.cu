/* Host side code. 
   Reduction of arbitrary sized vectors using atomics. 
   Also shows the use of pinned memory to map a portion of the CPU address space to the GPU's address space.

   Author: Naga Kandasamy
   Date created: February 14, 2017
   Date modified: May 17, 2020
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <sys/time.h>

/* Include kernel code */
#include "vector_reduction_kernel.cu"

void run_test(int);
double compute_on_device(float *, int);
void check_for_error(const char *);
extern "C" double compute_gold(float *, int);

int main(int argc, char **argv) 
{
    if (argc < 2) {
		printf("Usage: %s num-elements\n", argv[0]);
        printf("num-elements: Number of elements to be reduced\n");
		exit(EXIT_FAILURE);	
	}

	int num_elements = atoi(argv[1]);
	run_test(num_elements);
	exit(EXIT_SUCCESS);
}

/* Perform reduction on the CPU and the GPU and compare results for correctness */ 
void run_test(int num_elements) 
{
    struct timeval start, stop;	

	/* Allocate memory on the CPU for the input vector */
	int vector_size = sizeof(float) * num_elements;
	float *A = (float *)malloc(vector_size);
		
	/* Randomly generate input data to be values between -.5 and +.5 */	
	printf("\nCreating a random vector with %d elements\n", num_elements);
	srand(time(NULL));
    int i;
	for (i = 0; i < num_elements; i++)
		A[i] = rand()/(float)RAND_MAX - .5;
		
	/* Reduce vector on CPU */
	printf("\nReducing the vector with %d elements on the CPU\n", num_elements);
	gettimeofday(&start, NULL);
	double reference = compute_gold(A, num_elements);
    gettimeofday(&stop, NULL);
    printf("Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));
    printf("Answer = %f\n", reference);

	/* Compute the result vector on the GPU */ 
	printf("\nReducing the vector with %d elements on the GPU\n", num_elements);
    gettimeofday(&start, NULL);
	double gpu_result = compute_on_device(A, num_elements);
    gettimeofday (&stop, NULL);
    printf("Execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));
	printf("Answer = %f\n", gpu_result);
	
	/* Cleanup memory */
	free(A);
	exit(EXIT_SUCCESS);
}

double compute_on_device(float *A_on_host, int num_elements)
{
	float *A_on_device = NULL;
	double *result_on_device = NULL;
    struct timeval start, stop;	

    gettimeofday(&start, NULL);
	
    /* Allocate space on GPU for vector and copy contents over */
	hipMalloc((void**)&A_on_device, num_elements * sizeof(float));
	hipMemcpy(A_on_device, A_on_host, num_elements * sizeof(float), hipMemcpyHostToDevice);

	/* Allocate space for result on GPU and initialize */
	hipMalloc((void**)&result_on_device, sizeof(double));
	hipMemset(result_on_device, 0.0f, sizeof(double));

	/* Allocate space for the lock on GPU and initialize it */
	int *mutex_on_device = NULL;
	hipMalloc((void **)&mutex_on_device, sizeof(int));
	hipMemset(mutex_on_device, 0, sizeof(int));

    gettimeofday(&stop, NULL);
    printf("Data transfer time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

 	/* Set up execution grid on GPU */
	dim3 thread_block(THREAD_BLOCK_SIZE, 1, 1); 
	dim3 grid(NUM_BLOCKS,1);
	
	/* Launch kernel */
    gettimeofday(&start, NULL);
	vector_reduction_kernel<<<grid, thread_block>>>(A_on_device, result_on_device, num_elements, mutex_on_device);
	hipDeviceSynchronize();

    gettimeofday(&stop, NULL);
    printf("Kernel execution time = %fs\n", (float)(stop.tv_sec - start.tv_sec +\
                (stop.tv_usec - start.tv_usec)/(float)1000000));

	check_for_error("KERNEL FAILURE");

	double sum;
	hipMemcpy(&sum, result_on_device, sizeof(double), hipMemcpyDeviceToHost);

	/* Free memory */
	hipFree(A_on_device);
	hipFree(result_on_device);
	hipFree(mutex_on_device);

	return sum;
}

void check_for_error (const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		printf("CUDA ERROR: %s (%s). \n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
} 
