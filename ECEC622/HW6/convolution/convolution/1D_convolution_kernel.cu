#include "hip/hip_runtime.h"
#ifndef _CONVOLUTION_KERNEL_H_
#define _CONVOLUTION_KERNEL_H_

/* The kernel is stored in GPU  global memory in this kernel implementation. 
 * The input vector is also stored in global memory. 
 */ 
__global__ void convolution_kernel_v1(float *N, float *result, float *kernel, 
                                      int num_elements, int kernel_width)
{
    /* Obtain the index of the thread within the execution grid */ 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    float sum = 0.0;
    
    if(i >= num_elements)
        return;
    
    int N_start_point = i - (kernel_width/2); 
    for (int j = 0; j < kernel_width; j++) {
        if ((j + N_start_point >= 0) && (j + N_start_point < num_elements))
            sum += N[j + N_start_point] * kernel[j];
    }
    
    result[i] = sum;
}

/* The kernel is stored in GPU constant memory in kernel_c. 
 * The input vector is stored in global memory. 
 */
__global__ void convolution_kernel_v2(float *N, float *result, 
                                       int num_elements, int kernel_width)
{
    /* Obtain the index of the thread within the grid */
    int i = blockIdx.x * blockDim.x + threadIdx.x;     
    float sum = 0.0;

    if(i >= num_elements)
        return;

    int N_start_point = i - (kernel_width/2); 
    for (int j = 0; j < kernel_width; j++) {
        if ((j + N_start_point >= 0) && (j + N_start_point < num_elements))
            sum += N[j + N_start_point] * kernel_c[j]; 
    }
    
    result[i] = sum;
}

/* Tiled convolution kernel using shared memory.
 * The kernel is stored in GPU constant memory in kernel_c.
*/
__global__ void convolution_kernel_tiled(float *N, float *result, 
                                         int num_elements, int kernel_width)
{
    __shared__ float N_s[THREAD_BLOCK_SIZE + MAX_KERNEL_WIDTH - 1];
    
    /* Obtain the index of the thread within the grid. */   
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    int half_width = kernel_width/2;
    /* Load the left halo elements from the previous tile. 
     * The number of halo elements will be half_width.
     */
    int left_halo_index = (blockIdx.x - 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x >= (blockDim.x - half_width)) {
        if (left_halo_index < 0) 
            N_s[threadIdx.x - (blockDim.x - half_width)] = 0.0;
        else
            N_s[threadIdx.x - (blockDim.x - half_width)] = N[left_halo_index];
    }
		  
    /* Load the center elements for the tile */
    if (i < num_elements)			 
        N_s[half_width + threadIdx.x] = N[i];
    else 
        N_s[half_width + threadIdx.x] = 0.0;

    /* Load the right halo elements from the next tile. 
     * The number of halo elements will again be half_width.
     */
    int right_halo_index = (blockIdx.x + 1) * blockDim.x + threadIdx.x;
    if (threadIdx.x < half_width) {
        if (right_halo_index >= num_elements) 
            N_s[threadIdx.x + (blockDim.x + half_width)] = 0.0;
        else
            N_s[threadIdx.x + (blockDim.x + half_width)] = N[right_halo_index];
    }
		  
    __syncthreads();

    /* Convolve kernel over input elements */
    float sum = 0.0;
    for (int j = 0; j < kernel_width; j++)
        sum += N_s[j + threadIdx.x] * kernel_c[j];
		  
    result[i] = sum;
}
#endif /* _CONVOLUTION_KERNEL_H_ */
