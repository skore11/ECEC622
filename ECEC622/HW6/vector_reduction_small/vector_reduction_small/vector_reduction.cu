#include "hip/hip_runtime.h"
/* Vector reduction example using shared memory. 
 * Works for small vectors that can be operated upon by a single thread block.
 
 * Build as follows: make clean && make
 * Execute as follows: ./vector_reduction

 * Author: Naga Kandasamy
 * Date modified: May 15, 2020
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>
#include <time.h>

#define NUM_ELEMENTS 1024

/* Include kernel */
#include "vector_reduction_kernel.cu"

void run_test(int);
extern "C" double compute_gold(float *, int);
double compute_on_device(float *, int);
void check_CUDA_error(const char *);

int main(int argc, char **argv) 
{    
    int num_elements = NUM_ELEMENTS;
    if (num_elements > 1024) {
        fprintf(stderr, "Input exceeds bounds\n");
        exit(EXIT_FAILURE); 
    }

	run_test(num_elements);
	
    exit(EXIT_SUCCESS);
}

void run_test(int num_elements) 
{
    int array_mem_size = sizeof(float) * num_elements;

	/* Allocate memory on host to store input data */
	float* h_data = (float *)malloc(array_mem_size);

	/* Initialize input data to be floating-point values between [-.5, +.5] */
	srand(time(NULL));
    int i;
	for (i = 0; i < num_elements; i++) 
		h_data[i] = rand()/(float)RAND_MAX - 0.5;

	/* Calculate reference solution */
    printf("Reducing vector on CPU\n");
	double reference = compute_gold(h_data, num_elements);    
    printf("Answer = %f\n", reference);

    /* Calculate solution on device */
    printf("Reducing vector on GPU\n");    
    float gpu_result = compute_on_device(h_data, num_elements);
    printf("Answer = %f\n", gpu_result);

	/* Check for correctness */
    float eps = 1e-6;
	if (fabsf((reference - gpu_result)/reference) <= eps) 
        printf("TEST PASSED\n");
    else
        printf("TEST FAILED\n");

	free(h_data);
    exit(EXIT_SUCCESS);
}

/* Reduce vector on device */
double compute_on_device(float* h_data, int num_elements)
{
	float *d_data; /* Pointer to device address holding array */
    double *d_result; /* Pointer to device address holding result */
   	int data_size = sizeof(float) * num_elements;

	/* Allocate memory on device for the array */
	hipMalloc((void**)&d_data, data_size);
	check_CUDA_error("Error allocating memory");
	/* Copy data from host memory to device memory */
	hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice);
	check_CUDA_error("Error copying host to device memory");

    /* Allocate memory on device to store the reduction result */
    hipMalloc((void **)&d_result, sizeof(double));
    check_CUDA_error("Error allocating memory");

	/* Set up execution grid and invoke kernel */
	dim3 threads(num_elements, 1, 1);
	dim3 grid(1, 1);

    printf("Using reduction kernel, version 1\n");
	vector_reduction_kernel_v1<<<grid, threads>>>(d_data, d_result, num_elements);
    check_CUDA_error("Error in kernel");

    printf("Using reduction kernel, version 2\n");
    vector_reduction_kernel_v2<<<grid, threads>>>(d_data, d_result, num_elements);
	check_CUDA_error("Error in kernel");

	/* Copy result from device to host */
    double h_result;
	hipMemcpy(&h_result, d_result, sizeof(double), hipMemcpyDeviceToHost);
	check_CUDA_error("Error copying host to device memory");

	/* Clean up device memory */
	hipFree(d_data);
    hipFree(d_result);
	check_CUDA_error("Error freeing memory");

    return h_result;
}

void check_CUDA_error(const char *msg)
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err) {
		fprintf(stderr, "CUDA ERROR: %s (%s).\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}						 
}
