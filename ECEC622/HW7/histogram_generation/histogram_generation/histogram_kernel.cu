#include "hip/hip_runtime.h"
#ifndef _HISTOGRAM_KERNEL_H_
#define _HISTOGRAM_KERNEL_H_

/* Each thread block generates a histogram within shared memory. 
   These histrograms are then accumulated into the global histogram data structure 
   stored in global memory.
   */
__global__ void histogram_kernel_fast(int *input_data, int *histogram, 
                                      int num_elements, int histogram_size)
{
    __shared__ unsigned int s[HISTOGRAM_SIZE];
	
    /* Initialize shared memory */ 
    if(threadIdx.x < histogram_size)
        s[threadIdx.x] = 0;
		
    __syncthreads();

    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
	
    while (offset < num_elements) {
        atomicAdd(&s[input_data[offset]], 1);
        offset += stride;
    }	  
	
    __syncthreads();

    /* Accumulate histogram in shared memory into global memory */
    if (threadIdx.x < histogram_size) 
        atomicAdd(&histogram[threadIdx.x], s[threadIdx.x]);
}

/* The shared histrogram data structure is stored in global memory and each thread directly 
   accumulates into the structure using an atomic operation. There is a lot of contention 
   between threads in this implementation. 
   */
__global__ void histogram_kernel_slow(int *input_data, int *histogram, 
                                      int num_elements, int histogram_size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; 
	
    while (offset < num_elements) {
        atomicAdd(&histogram[input_data[offset]], 1);
        offset += stride;
    }	
}

#endif /* _HISTOGRAM_KERNEL_H_ */
