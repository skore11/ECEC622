#include "hip/hip_runtime.h"
/* Host side code that calls a GPU kernel to perform vector addition on the GPU using a single thread block.
   
   We restrict the size of the vector to be up to 1024 elements which is the maximum thread block size on this 
   GPU.

    Author: Naga Kandasamy
    Date modified: May 3, 2020
*/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

#define NUM_ELEMENTS 1024

/* Include the kernel code during the compiler preprocessing step */
#include "vector_addition_kernel.cu"

void run_test(void);
void compute_on_device(float *, float *, float *, int);
extern "C" void compute_gold(float *, float *, float *, int);

int main(int argc, char **argv) 
{
    run_test();
	
    exit(EXIT_SUCCESS);
}

/* Perform vector addition on the CPU and the GPU */
void run_test(void) 
{                                                        
    int num_elements = NUM_ELEMENTS;
	float diff;
	int i; 

    /* Allocate memory on the CPU for input vectors A and B, and output vector C */
    int vector_length = sizeof(float) * num_elements;
	float *A = (float *)malloc(vector_length);
	float *B = (float *)malloc(vector_length);
	float *gold_result = (float *)malloc(vector_length);            /* Result vector computed on CPU */
	float *gpu_result = (float *)malloc(vector_length);             /* Result vector computed on GPU */
	
	/* Initialize the input data to be integer values between 0 and 5 */ 
	for (i = 0; i < num_elements; i++) {
		A[i] = floorf(5 * (rand() / (float)RAND_MAX));
     	B[i] = floorf(5 * (rand() / (float)RAND_MAX));
	}

	/* Compute reference solution on CPU */
	compute_gold(A, B, gold_result, num_elements);
    
	/* Compute result vector on GPU */ 
	compute_on_device(A, B, gpu_result, num_elements);

	/* Compute differences between CPU and GPU results */
    diff = 0.0;
    for (i = 0; i < num_elements; i++)
		diff += fabsf(gold_result[i] - gpu_result[i]);

	printf("Difference between the CPU and GPU result = %f\n", diff);
   
	/* Cleanup memory */
	free((void *)A);
	free((void *)B);
	free((void *)gold_result);
	free((void *)gpu_result);
	
	return;
}

/* Vector addition on GPU */
void compute_on_device(float *A_on_host, float *B_on_host, float *gpu_result, int num_elements)
{                                                                                                 
	float *A_on_device = NULL;
	float *B_on_device = NULL;
	float *C_on_device = NULL; 

	/* Allocate space on GPU for vectors A and B, and copy contents of vectors to GPU */
	hipMalloc((void**)&A_on_device, num_elements * sizeof(float));
	hipMemcpy(A_on_device, A_on_host, num_elements * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void**)&B_on_device, num_elements * sizeof(float));
	hipMemcpy(B_on_device, B_on_host, num_elements * sizeof(float), hipMemcpyHostToDevice);

	/* Allocate space for result vector on GPU */
	hipMalloc((void**)&C_on_device, num_elements * sizeof(float));
	
 	/* Set up execution grid on the GPU */
	dim3 thread_block(num_elements, 1, 1);          /* Set number of threads in thread block */
	dim3 grid(1,1);

	vector_addition_kernel<<<grid, thread_block>>>(A_on_device, B_on_device, C_on_device, num_elements);                                                                                                       
	/* Copy result vector back from GPU */ 
	hipMemcpy(gpu_result, C_on_device, num_elements * sizeof(float), hipMemcpyDeviceToHost);
	
	/* Free memory on GPU */
	hipFree(A_on_device);
	hipFree(B_on_device);
	hipFree(C_on_device);
}
     
